// Question 1


#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <ctype.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#include <unistd.h>

// Number of threads in one block (possible range is 32...1024):
#define blockSize 256

// Total number of threads (total number of elements to process in the kernel):
#define N 256 * 128

// Maximum value of distance
#define MAX_DIST 1.42

#define MIN(X, Y) (((X) < (Y)) ? (X) : (Y))
#define SQUARE(X) ((X) * (X))

// Input array (global host memory):
float X[N];
float Y[N];

__device__ float d_X[N];
__device__ float d_Y[N];
__device__ float d_min_k1;
__device__ float d_min_k2;

//++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
static inline __device__ float fatomicMin(float *addr, float value)
{
    float old = *addr, assumed;
    if (old <= value)
        return old;
    do
    {
        assumed = old;
        old = atomicCAS((unsigned int *)addr, __float_as_int(assumed), __float_as_int(value));
    } while (old != assumed);

    return old;
}

__global__ void init_kernel()
{
    d_min_k1 = MAX_DIST;
    d_min_k2 = MAX_DIST;

    return;
}

// one tread per particle
__global__ void OneThreadPerParticleKernel()
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    float thread_min = MAX_DIST, next_dist = MAX_DIST;

    // Calculate the distance for the current thread
    for (int j = i + 1; j < N; j++)
    {
        next_dist = sqrtf(SQUARE(d_X[i] - d_X[j]) + SQUARE(d_Y[i] - d_Y[j]));
        thread_min = fminf(next_dist, thread_min);
    }

    // Warp-level reduction
    unsigned int mask = __ballot_sync(0xFFFFFFFF, true); // Mask for all threads in the warp
    float min_value = thread_min;

    // Perform reduction within the warp
    for (int offset = warpSize / 2; offset > 0; offset >>= 1)
    {
        float neighbor = __shfl_down_sync(mask, min_value, offset);
        min_value = fminf(min_value, neighbor);
    }

    // Use the first thread of each warp to write the result to global memory
    if (threadIdx.x % warpSize == 0)
    {
        // Use atomic operation to update the global minimum
        fatomicMin(&d_min_k1, min_value);
    }
}


__global__ void OneThreadPerPairKernel()
{
    int k = threadIdx.x + blockDim.x * blockIdx.x;
   
    int row = k / (N - 1);
    int col = row + 1 + (k % (N - 1));

    int i = (col > N - 1) ? N - 1 - row : row;
    int j = (col > N - 1) ? col - row : col;

    // Calculate the distance for the current pair
    float distance = sqrtf(SQUARE(d_X[i] - d_X[j]) + SQUARE(d_Y[i] - d_Y[j]));

    // Warp-level reduction
    unsigned int mask = __ballot_sync(0xFFFFFFFF, true); // Mask indicating all threads in the warp
    float min_value = distance;

    // Perform reduction within the warp
    for (int offset = warpSize / 2; offset > 0; offset >>= 1) {
        float neighbor = __shfl_down_sync(mask, min_value, offset);
        min_value = fminf(min_value, neighbor);
    }

    // Use the first thread of the warp to write the result to global memory
    if (threadIdx.x % warpSize == 0) {
        // Use atomic operation to update global minimum
        fatomicMin(&d_min_k2, min_value);
    }
}


int main(int argc, char **argv)
{
    double min0;
    float next_dist, min_k1, min_k2;
    int numBlocks;

    // Initializing random number generator:
    srand(235829410);

    // Initializing the input array:
    for (int i = 0; i < N; i++)
    {
        X[i] = (float)rand() / (float)RAND_MAX;
        Y[i] = (float)rand() / (float)RAND_MAX;
    }

    //----------------------------------CPU-------------------------------------------
    // time_t begin_cpu, end_cpu;

    clock_t begin_cpu = clock();
    // time(&begin_cpu);

    min0 = MAX_DIST;
    next_dist = MAX_DIST;
    for (int i = 0; i < N; i++)
    {
        for (int j = i + 1; j < N; j++)
        {
            next_dist = sqrt(SQUARE(X[i] - X[j]) + SQUARE(Y[i] - Y[j]));
            min0 = (next_dist < min0) ? next_dist : min0;
        }
    }

    clock_t end_cpu = clock();
    // time(&end_cpu);

    double cpu_time = double(end_cpu - begin_cpu)/CLOCKS_PER_SEC;

    printf("------------------------- N  =  %d ------------------------- \n", N);

    printf("---------------------------------------\n");
    printf("Serial code on CPU\n");
    printf("CPU Time: %.7f ms \n", cpu_time * 1000);
    printf("CPU distance: %.7f \n", min0);
    printf("---------------------------------------\n");

    //------------------------------------INIT KERNELS-----------------------------------

    hipMemcpyToSymbol(HIP_SYMBOL(d_X), X, N * sizeof(float), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_Y), Y, N * sizeof(float), 0, hipMemcpyHostToDevice);

    init_kernel<<<1, 1>>>();
    hipDeviceSynchronize();

    //------------------------------------GPU 2-----------------------------------------
    hipEvent_t start, stop;
    float kernel_timer;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    numBlocks = N / blockSize;
    OneThreadPerParticleKernel<<<numBlocks, blockSize>>>();

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&kernel_timer, start, stop);

    hipMemcpyFromSymbol(&min_k1, HIP_SYMBOL(d_min_k1), sizeof(float), 0, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    printf("--------------------------------------\n");
    printf("One thread per particle \n");
    printf("GPU kernel 1 took %.7f ms\n", kernel_timer);
    printf("GPU kernel 1 distance: %.7f \n", min_k1);
    printf("--------------------------------------\n");

    //------------------------------------GPU 2----------------------------------------
    hipEvent_t start_2, stop_2;
    float kernel_timer_2;

    hipEventCreate(&start_2);
    hipEventCreate(&stop_2);
    hipEventRecord(start_2, 0);

    numBlocks = N / blockSize / 2 * (N - 1);
    OneThreadPerPairKernel<<<numBlocks, blockSize>>>();

    hipEventRecord(stop_2, 0);
    hipEventSynchronize(stop_2);
    hipEventElapsedTime(&kernel_timer_2, start_2, stop_2);

    hipMemcpyFromSymbol(&min_k2, HIP_SYMBOL(d_min_k2), sizeof(float), 0, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    printf("----------------------------------------\n");
    printf("One thread per pair\n");
    printf("GPU kernel 2 took %.7f ms\n", kernel_timer_2);
    printf("GPU kernel 2 distance: %.7f \n", min_k2);
    printf("----------------------------------------\n");

    return 0;
}