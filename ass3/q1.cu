// Question 1


#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <ctype.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#include <unistd.h>

// Number of threads in one block (possible range is 32...1024):
#define blockSize 256

// Total number of threads (total number of elements to process in the kernel):
#define N 256 * 128

// Maximum value of distance
#define MAX_DIST 1.42

#define MIN(X, Y) (((X) < (Y)) ? (X) : (Y))
#define SQUARE(X) ((X) * (X))

// Input array (global host memory):
float X[N];
float Y[N];

__device__ float d_X[N];
__device__ float d_Y[N];
__device__ float d_min_k1;
__device__ float d_min_k2;

//++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
static inline __device__ float fatomicMin(float *addr, float value)
{
    float old = *addr, assumed;
    if (old <= value)
        return old;
    do
    {
        assumed = old;
        old = atomicCAS((unsigned int *)addr, __float_as_int(assumed), __float_as_int(value));
    } while (old != assumed);

    return old;
}

__global__ void init_kernel()
{
    d_min_k1 = MAX_DIST;
    d_min_k2 = MAX_DIST;

    return;
}

// one tread per particle
__global__ void OneThreadPerParticleKernel()
{

    __shared__ float shared_min[blockSize];

    int i = threadIdx.x + blockDim.x * blockIdx.x;
    float thread_min = MAX_DIST, next_dist = MAX_DIST;

    // calculate the distance related to i and find the min of current thread
    for (int j = i + 1; j < N; j++)
    {
        next_dist = sqrt(SQUARE(d_X[i] - d_X[j]) + SQUARE(d_Y[i] - d_Y[j]));
        thread_min = (next_dist < thread_min) ? next_dist : thread_min;
    }
    shared_min[threadIdx.x] = thread_min;

    // To make sure all threads finished calc
    __syncthreads();

    // find the min within the block
    int nTotalThreads = blockDim.x; // Total number of active threads;
    // only the first half of the threads will be active.
    while (nTotalThreads > 1)
    {
        int halfPoint = (nTotalThreads >> 1); // divide by two
        if (threadIdx.x < halfPoint)
        {
            int thread2 = threadIdx.x + halfPoint;
            shared_min[threadIdx.x] = MIN(shared_min[threadIdx.x], shared_min[thread2]); // Pairwise summation
        }
        __syncthreads();
        nTotalThreads = halfPoint; // Reducing the binary tree size by two
    }

    // find the min among blocks, i.e. global min
    if (threadIdx.x == 0)
    {
        fatomicMin(&d_min_k1, shared_min[0]);
    }

    return;
}

__global__ void OneThreadPerPairKernel()
{
    __shared__ float shared_min[blockSize];

    int k = threadIdx.x + blockDim.x * blockIdx.x;
   
    int row = k / (N - 1);
    int col = row + 1 + (k % (N - 1));

    int i = (col > N - 1) ? N - 1 - row : row;
    int j = (col > N - 1) ? col - row : col;

    // calculate the distance related to i and find the min of current thread
    shared_min[threadIdx.x] = sqrt(SQUARE(d_X[i] - d_X[j]) + SQUARE(d_Y[i] - d_Y[j]));

    // To make sure all threads finished calc
    __syncthreads();

    // find the min within the block
    int nTotalThreads = blockDim.x; // Total number of active threads;
    // only the first half of the threads will be active.
    while (nTotalThreads > 1)
    {
        int halfPoint = (nTotalThreads >> 1); // divide by two
        if (threadIdx.x < halfPoint)
        {
            int thread2 = threadIdx.x + halfPoint;
            shared_min[threadIdx.x] = MIN(shared_min[threadIdx.x], shared_min[thread2]); // Pairwise summation
        }
        __syncthreads();
        nTotalThreads = halfPoint; // Reducing the binary tree size by two
    }

    // find the min among blocks, i.e. global min
    if (threadIdx.x == 0)
    {
        fatomicMin(&d_min_k2, shared_min[0]);
    }

    return;
}

int main(int argc, char **argv)
{
    double min0;
    float next_dist, min_k1, min_k2;
    int numBlocks;

    // Initializing random number generator:
    srand(235829410);

    // Initializing the input array:
    for (int i = 0; i < N; i++)
    {
        X[i] = (float)rand() / (float)RAND_MAX;
        Y[i] = (float)rand() / (float)RAND_MAX;
    }

    //----------------------------------CPU-------------------------------------------
    // time_t begin_cpu, end_cpu;

    clock_t begin_cpu = clock();
    // time(&begin_cpu);

    min0 = MAX_DIST;
    next_dist = MAX_DIST;
    for (int i = 0; i < N; i++)
    {
        for (int j = i + 1; j < N; j++)
        {
            next_dist = sqrt(SQUARE(X[i] - X[j]) + SQUARE(Y[i] - Y[j]));
            min0 = (next_dist < min0) ? next_dist : min0;
        }
    }

    clock_t end_cpu = clock();
    // time(&end_cpu);

    double cpu_time = double(end_cpu - begin_cpu)/CLOCKS_PER_SEC;

    printf("------------------------- N  =  %d ------------------------- \n", N);

    printf("---------------------------------------\n");
    printf("Serial code on CPU\n");
    printf("CPU Time: %.7f ms \n", cpu_time * 1000);
    printf("CPU distance: %.7f \n", min0);
    printf("---------------------------------------\n");

    //------------------------------------INIT KERNELS-----------------------------------

    hipMemcpyToSymbol(HIP_SYMBOL(d_X), X, N * sizeof(float), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_Y), Y, N * sizeof(float), 0, hipMemcpyHostToDevice);

    init_kernel<<<1, 1>>>();
    hipDeviceSynchronize();

    //------------------------------------GPU 2-----------------------------------------
    hipEvent_t start, stop;
    float kernel_timer;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    numBlocks = N / blockSize;
    OneThreadPerParticleKernel<<<numBlocks, blockSize>>>();

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&kernel_timer, start, stop);

    hipMemcpyFromSymbol(&min_k1, HIP_SYMBOL(d_min_k1), sizeof(float), 0, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    printf("--------------------------------------\n");
    printf("One thread per particle \n");
    printf("GPU kernel 1 took %.7f ms\n", kernel_timer);
    printf("GPU kernel 1 distance: %.7f \n", min_k1);
    printf("--------------------------------------\n");

    //------------------------------------GPU 2----------------------------------------
    hipEvent_t start_2, stop_2;
    float kernel_timer_2;

    hipEventCreate(&start_2);
    hipEventCreate(&stop_2);
    hipEventRecord(start_2, 0);

    numBlocks = N / blockSize / 2 * (N - 1);
    OneThreadPerPairKernel<<<numBlocks, blockSize>>>();

    hipEventRecord(stop_2, 0);
    hipEventSynchronize(stop_2);
    hipEventElapsedTime(&kernel_timer_2, start_2, stop_2);

    hipMemcpyFromSymbol(&min_k2, HIP_SYMBOL(d_min_k2), sizeof(float), 0, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    printf("----------------------------------------\n");
    printf("One thread per pair\n");
    printf("GPU kernel 2 took %.7f ms\n", kernel_timer_2);
    printf("GPU kernel 2 distance: %.7f \n", min_k2);
    printf("----------------------------------------\n");

    return 0;
}